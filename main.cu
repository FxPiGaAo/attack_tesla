#include<stdio.h>
#include<iostream>
#include<malloc.h>
#include<ctime>
#include<hip/hip_runtime.h>
#include<assert.h>
using namespace std;
//__constant__ int* device_array;

__global__ void test_clock(int &delay, int &add){
   int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
   clock_t start=0;
   if(threadID == 0) start = clock();
   for(int k=0;k<100;k++){
      for(int j =0;j<10;j++){
         for(int i=0;i<100;i++){
            if(threadID==0){add+=i;}
	    //add+=j;
	    //if(threadID<11){add+=k;}
		 add+=k;
         }
      }
   }
   if(threadID==0){clock_t end = clock();
   delay = (int)(end - start);}
}



__global__ void sequence_read(long long int &latency, int* device_array, int n, int access_number){
   extern __shared__ int shared_array[];
   for(int i=0;i<n;i++){shared_array[i]=device_array[i];}
   int* j = &shared_array[0];
   //for(int i=0;i<access_number;i++){j=*(int **)j;}
   //j = &shared_array[0];
   long long int temp = clock64();
   for(int i=0;i<access_number;i++){j=*(int **)j;}
   latency = clock64() - temp;
}
__global__ void static_sequence_read(int &latency, long long unsigned* device_array, int access_number, long long unsigned &last_access_value){
   int threadID = (blockIdx.x * blockDim.x) + threadIdx.x;
   //__shared__ int shared_array[64];
   //__constant__ int shared_array[64];
   //for(int i=0;i<64;i++){shared_array[i]=device_array[i];}
   long long unsigned *j;
   if(threadID == 0){
       j =&device_array[0];
       //int* j = &shared_array[0];
       for(int i=0;i<access_number;i++){j=*(long long unsigned **)j;}
   }
   //j = &shared_array[0];
   clock_t temp=0;
   if(threadID == 0){temp = clock();}
   //for(int i=0;i<access_number;i++){if(threadID <5) j=*(int**)j;}
   //long long int temp = clock64();
   for(int i=0;i<access_number;i++){if(threadID == 0) j=*(long long unsigned **)j;}
   if(threadID == 0){
	   latency = (int)(clock() - temp);
	   last_access_value = j[0];
   }
}
int main(void){/*
   for(int array_size = 64; array_size<2048;array_size+=8){
     int device_size = sizeof(int)*array_size;
     int* device_array;
     int* host_array = (int*)malloc(array_size*sizeof(int*));
     cudaMalloc((void**)&device_array,device_size);
     int stride = 4;
     for(int i = 0; i < array_size; i++){
         int t = i + stride;
         if(t >= array_size) t %= stride;
         host_array[i] = *((int*)(&device_array)) + 4*t;//converse the device from int* to int; 4 is the byte size of an int type
     }
     long long int* timing = (long long int*)malloc(sizeof(long long int));
     long long int* timing_d;
     cudaMalloc((void**)&timing_d, sizeof(long long int));
     printf("start computing!\n");
     cudaMemcpy(device_array,host_array,device_size,cudaMemcpyHostToDevice);
     sequence_read<<<1,1,array_size*sizeof(int)>>>(timing_d[0], device_array, array_size, 1000000);
     cudaMemcpy(timing,timing_d,sizeof(long long int),cudaMemcpyDeviceToHost);
     printf ("It took me %lld clicks.\n",timing[0]);
     delete host_array;
     //printf ("It took me %Lf clicks.\n",timing[0]);
   }
*/
	/*
	//cudaEvent_t event1, event2;
	//cudaEventCreate(&event1);
	//cudaEventCreate(&event2);
	
	int* d_time;
	int time;
     int add = 0;
     int* d_add;printf("%d,%d\n",time,add);
     cudaMalloc((void**)&d_time,sizeof(int));
     cudaMalloc((void**)&d_add,sizeof(int));
     cudaMemcpy(d_add,&add,sizeof(int),cudaMemcpyHostToDevice); 
     clock_t start = clock();
     //cudaEventRecord(event1 ,0);
     test_clock<<<1,1>>>(d_time[0],d_add[0]);
     //cudaEventRecord(event2,0);
     //cudaEventSynchronize(event1);
     //cudaEventSynchronize(event2);
     //cudaDeviceSynchronize();
     clock_t end = clock();
     cudaMemcpy(&time,d_time,sizeof(int),cudaMemcpyDeviceToHost);
     cudaMemcpy(&add,d_add,sizeof(int),cudaMemcpyDeviceToHost);
     long double time_elapsed_ms = 1000.0 * (end-start) / CLOCKS_PER_SEC;
     cout << "CPU time used: " << time_elapsed_ms << " ms\n";
     printf("%d,%d\n",time,add);
     //float dt_ms;
     //cudaEventElapsedTime(&dt_ms, event1, event2);
     //cout << "cuda event elpased time:" << dt_ms << " ms\n";
*/

     long long unsigned array_size = 16;
     long long unsigned device_size = sizeof(long long unsigned)*array_size;
     long long unsigned* device_array;
     long long unsigned* host_array = (long long unsigned*)malloc(array_size*sizeof(long long unsigned*));
     assert(hipSuccess == hipMalloc((void**)&device_array,device_size));
     int stride = 4;
     for(int i = 0; i < array_size; i++){
         int t = i + stride;
         if(t >= array_size) t %= stride;
         host_array[i] = *((long long unsigned*)(&device_array)) + 4*t;//converse the device from int* to int; 4 is the byte size of an int type
     }

/*
     cout<< "sizeof long long unsigned" << sizeof(long long unsigned) << endl;
     cout<< "device array adress: " << (long long unsigned)device_array << endl;
     for(int i=0;i<array_size;i++){
         cout << host_array[i] << endl;
     }
     return 0;
*/


     int* timing = (int*)malloc(sizeof(int));
     int* timing_d;
     printf ("It took me %d clicks before the funvtion call.\n",timing[0]);
     assert(hipSuccess == hipMalloc((void**)&timing_d, sizeof(int)));
     long long unsigned* last_access_value = (long long unsigned*)malloc(sizeof(long long unsigned));
     long long unsigned* d_last_access_value;
     printf ("original last_access value: %llu\n", last_access_value[0]);
     assert(hipSuccess == hipMalloc((void**)&d_last_access_value, sizeof(long long unsigned)));
     printf("start computing!\n");
     assert(hipSuccess == hipMemcpy(device_array,host_array,device_size,hipMemcpyHostToDevice));

     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 4, d_last_access_value[0]);
     assert(hipSuccess == hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost));
     assert(hipSuccess == hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost));
     hipDeviceSynchronize();
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);


     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 4, d_last_access_value[0]);
     hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost);
     hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost);
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0],last_access_value[0]);

     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 4, d_last_access_value[0]);
     hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost);
     hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost);
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);

     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 4, d_last_access_value[0]);
     hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost);
     hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost);
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);

     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 10000, d_last_access_value[0]);
     hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost);  
     hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost);
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);
     
     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 2000, d_last_access_value[0]);
     hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost);
     hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost);
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);

     static_sequence_read<<<1,32>>>(timing_d[0], device_array, 1, d_last_access_value[0]);
     hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned),hipMemcpyDeviceToHost);
     hipMemcpy(timing,timing_d,sizeof(int),hipMemcpyDeviceToHost);
     printf ("It took me %d clicks, last_access value: %llu.\n",timing[0], last_access_value[0]);

     delete host_array;
     //printf ("It took me %Lf clicks.\n",timing[0]);

   return 0;
} 





