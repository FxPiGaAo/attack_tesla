
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<malloc.h>

__global__ void sequence_read(long long int &latency, int* device_array, int n, int access_number){
   extern __shared__ int shared_array[];
   for(int i=0;i<n;i++){shared_array[i]=device_array[i];}
   int* j = &shared_array[0];
   //for(int i=0;i<access_number;i++){j=*(int **)j;}
   //j = &shared_array[0];
   long long int temp = clock64();
   for(int i=0;i<access_number;i++){j=*(int **)j;}
   latency = clock64() - temp;
}
int main(void){
   for(int array_size = 64; array_size<2048;array_size+=8){
     int device_size = sizeof(int)*array_size;
     int* device_array;
     int* host_array = (int*)malloc(array_size*sizeof(int*));
     hipMalloc((void**)&device_array,device_size);
     int stride = 4;
     for(int i = 0; i < array_size; i++){
         int t = i + stride;
         if(t >= array_size) t %= stride;
         host_array[i] = *((int*)(&device_array)) + 4*t;//converse the device from int* to int; 4 is the byte size of an int type
     }
     long long int* timing = (long long int*)malloc(sizeof(long long int));
     long long int* timing_d;
     hipMalloc((void**)&timing_d, sizeof(long long int));
     printf("start computing!\n");
     hipMemcpy(device_array,host_array,device_size,hipMemcpyHostToDevice);
     sequence_read<<<1,1,array_size*sizeof(int)>>>(timing_d[0], device_array, array_size, array_size/stride);
     hipMemcpy(timing,timing_d,sizeof(long long int),hipMemcpyDeviceToHost);
     printf ("It took me %lld clicks.\n",timing[0]);
     delete host_array;
     //printf ("It took me %Lf clicks.\n",timing[0]);
   }



/*
   for(int array_size = 64; array_size<2048;array_size+=8){
     int device_size = sizeof(int)*array_size;
     int* device_array;
     int* host_array = (int*)malloc(array_size*sizeof(int*));
     cudaMalloc((void**)&device_array,device_size);
     int stride = 4;
     for(int i = 0; i < array_size; i++){
         int t = i + stride;
         if(t >= array_size) t %= stride;
         host_array[i] = *((int*)(&device_array)) + 4*t;//converse the device from int* to int; 4 is the byte size of an int type
     }
     long long int* timing = (long long int*)malloc(sizeof(long long int));
     long long int* timing_d;
     cudaMalloc((void**)&timing_d, sizeof(long long int));
     printf("start computing!\n");
     cudaMemcpy(device_array,host_array,device_size,cudaMemcpyHostToDevice);
     sequence_read<<<1,1>>>(timing_d[0], device_array, 1);
     cudaMemcpy(timing,timing_d,sizeof(long long int),cudaMemcpyDeviceToHost);
     printf ("It took me %lld clicks.\n",timing[0]);
     delete host_array;
     //printf ("It took me %Lf clicks.\n",timing[0]);
   }
*/
   return 0;
} 





