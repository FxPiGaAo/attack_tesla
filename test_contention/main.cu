#include<stdio.h>
#include<iostream>
#include<malloc.h>
#include<ctime>
#include<hip/hip_runtime.h>
#include<assert.h>
using namespace std;
__global__ void loop_stride_access(int* latency, long long unsigned* device_array, int access_number, long long unsigned* last_access_value, int array_size){
   int threadx =threadIdx.x;
   int smid = blockIdx.x;
   clock_t start, end;
   long long unsigned *j;
   j = &(device_array[array_size*smid]);
   for(int i=0;i<access_number;i++){if(threadx == 0) j=*(long long unsigned **)j;}//first acces to cache the data
   if(threadx == 0) last_access_value[smid] = j[0];
   j = &(device_array[array_size*smid]);
   __syncthreads();//finish intializing the array
   if(threadx == 0){
	   start = clock();
	   for(int k=0;k<100;k++){//do the same thing 100 times to increase the access time difference
           	for(int i=0;i<access_number;i++){j=*(long long unsigned **)j;}//access the data array
		last_access_value[smid] = j[0];
		j = &(device_array[array_size*smid]);
	   }
	   end = clock();
	   latency[smid] = (int)(end - start);
	   last_access_value[smid] = j[0];
   }
}


int main(void){
     


     for(int stride = 16;stride<1024;stride+=16){

         long long unsigned array_size = 8192;//let the array overflow the l1 cache;array_size = 64KB/8byte = 8192
         int sm_max = 20;
         printf("%d\t",stride);
         long long unsigned device_size = sizeof(long long unsigned)*array_size*sm_max;
         long long unsigned* device_array;
         long long unsigned* host_array = (long long unsigned*)malloc(array_size*sizeof(long long unsigned*)*sm_max);
         assert(hipSuccess == hipMalloc((void**)&device_array,device_size));
         for(int sm_id =0;sm_id<sm_max;sm_id++){
             for(int i = 0; i < array_size; i++){
                 int t = i + stride;
                 if(t >= array_size) t %= stride;
                 host_array[i+array_size*sm_id] = (long long unsigned)(&(device_array[sm_id*array_size])) + sizeof(long long unsigned)*t;//converse the device from int* to int; 4 is the byte size of an int type
             }
         }
   

         int* timing = (int*)malloc(sizeof(int)*sm_max);
         int* timing_d;
         assert(hipSuccess == hipMalloc((void**)&timing_d, sizeof(int)*sm_max));
         long long unsigned* last_access_value = (long long unsigned*)malloc(sizeof(long long unsigned)*sm_max);
         long long unsigned* d_last_access_value;
         assert(hipSuccess == hipMalloc((void**)&d_last_access_value, sizeof(long long unsigned)*sm_max));
         assert(hipSuccess == hipMemcpy(device_array,host_array,device_size,hipMemcpyHostToDevice));
    
         double access_time;


         hipDeviceSynchronize();
         loop_stride_access<<<sm_max,1>>>(timing_d, device_array, 48, d_last_access_value, array_size);
         hipDeviceSynchronize();
         assert(hipSuccess == hipMemcpy(timing,timing_d,sizeof(int)*sm_max,hipMemcpyDeviceToHost));
         assert(hipSuccess == hipMemcpy(last_access_value,d_last_access_value,sizeof(long long unsigned)*sm_max,hipMemcpyDeviceToHost));
         hipDeviceSynchronize();
         access_time = 0;
         for(int i=0;i<sm_max;i++){
             access_time+=timing[i];
         }
         printf("%lf\n",access_time/sm_max);
      

         delete host_array;
         delete timing;
         delete last_access_value;


     }
     return 0;
} 





